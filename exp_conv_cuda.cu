#include "hip/hip_runtime.h"
//

//  test_cuda.c

//

//

//  Created by Andrew Christlieb on 3/3/14.

//  This code is a CUDA kernal for domain decompositon

//  of the implcit maxwell solver.

//  The code is mearly a test code and use #define

//  where the varables should be done with ArgV ArgC.

//

//  This is mealy a 1D test code!  NOT GREAT, but menat

//  for learning!

//


//GPU Version: parallel add of arbitrary length.

#include <stdio.h>

#include <stdbool.h>

#include <math.h>

#include <hip/hip_runtime.h>


const int N = (int)pow(2,27);    /* num grid point */

const int M = 16;      /* max number mesh cells */



//each tid - denotes a subdominant up to M cells

//

//   |----------|----------|----------|...----------|-----| (note: last cell<M)

//     domain 0   domain 1   domain 2     domain K-2  domain K-1

//


/* 
Function name: debug_tool_output_to_file

Function type: host function

Purpose: Given a pointer to an array of single-digit integers of length N, outputs the array to a text file.
			The text file is displayed in rows of M entries.

Inputs: int * debugArray_ptr - a pointer to an array of single digit integers
			const int N - the length of the array
			const int M - the length of a row in the text file

Outputs: generates a file named debug_output.txt
*/


void debug_tool_output_to_file(int * debugArray_ptr, const int N, const int M){
    int k = N/M;           /* number of whole sub domains */

    int k_end = N%M;         /* number of points in last domain */

    int k_tot;             /* total number of sub domains */

    int loop_over_y_cells; /* number of mesh point in domain */

    int cell_index,i,j;     /* cells_index is the flattend index of the cell */

                           /* j is merely a counter */

    bool test;             /* flag to indicate if we have a single nonuniform

                               sub domain */

	char output;

    /* set up logic for when number of subdomains

     does not evenly divide total number of cells in a line. */

    if(k_end>0)

    {

        k_tot = k+1;       /* number of sub domains */

        test = true;       /* flag for indicating a single special domain */

    }

    else

    {

        k_tot = k;         /* number of sub domains */

        test = false;

    }
	
	
	FILE * fp;
	
	fp = fopen("debug_output.txt","w");
	
	for(i = 0; i< k_tot; i++){
		if((i==(k_tot-1))&&test){
			loop_over_y_cells = k_end;
		}
		else{
			loop_over_y_cells = M;
		}
		for(j = 0; j< loop_over_y_cells; j++ ){
			
			cell_index = i*M+j;
			output = (char)(((int)'0')+debugArray_ptr[cell_index]);
			fputc((int)output,fp);
		}
		fprintf(fp,"\n");
	}
	
	fclose(fp);
	
}


/*
Function name: localWeightAndSweep_L

Function type: CUDA kernel

Purpose: Performs the local deposit step then the local sweep step over each subdomain for the left integral part.

Inputs: float * JL_d - pointer to array on device storing the local integral values of the left part
		float * val_d - pointer to array on device storing the integrand values
		const int N - number of grid points in the array
		const int M - number of grid points in one subdomain
		float nu - numerical parameter equal to dx*alpha
		int * debugArray_d - pointer to array of single-digit integers on device; used for diagnostic purposes only

Outputs: JL_d is written to at all grid points
*/
__global__ void localWeightAndSweep_L(float *JL_d, float *val_d, const int N, const int M, float nu, int *debugArray_d)

{

    int k = N/M;           /* number of whole sub domains */

    int k_end = N%M;         /* number of points in last domain */

    int k_tot;             /* total number of sub domains */

    int loop_over_y_cells; /* number of mesh point in domain */

    int cell_index,j;     /* cells_index is the flattend index of the cell */

                           /* j is merely a counter */

    bool test;             /* flag to indicate if we have a single nonuniform

                               sub domain */
		

	int startLoopIndex; /* Loop start index in subdomain - should be 1 in first subdomain, 0 in others. */
	int endLoopIndex; /* Loop end index - should be k_end-1 in the last subdomain, M in others. */ 
		


	float ex = exp(-nu);


		/* Quadrature weights */
	float P = 1.0 - (1.0-ex)/nu;
	float Q = -ex+(1.0-ex)/nu;
	float R = (1.0-ex)/(nu*nu)-(1.0+ex)/(2.0*nu);



    /* set up logic for when number of subdomains

     does not evenly divide total number of cells in a line. */

    if(k_end>0)

    {

        k_tot = k+1;       /* number of sub domains */

        test = true;       /* flag for indicating a single special domain */

    }

    else

    {

        k_tot = k;         /* number of sub domains */

        test = false;

    }



    /* use unique tid to identify which sub domain */

    int tid = threadIdx.x+blockIdx.x*blockDim.x;  //index of sub domain



    /* let the kernel address more than a single sub domain */

    while(tid<k_tot){
		
		/* If we are in the first subdomain, the first point needs special treatment, as the usual quadrature stencil will extend outside of the domain. */
		/* We have that JL[0]=0. */
		if(tid==0)
		{
			startLoopIndex = 1;
			JL_d[0]=0.0;
		}
		else
		
			startLoopIndex = 0;

		
        if((tid==k_tot-1)&&(test))

            loop_over_y_cells= k_end;

        else

            loop_over_y_cells= M;

		
		/* If we are in the last subdomain, the last point needs special treatment, as the usual quadrature stencil will extend outside of the domain. */
		/* We have that JL[N-1] is NOT zero, so we need special treatment at the right endpoint for JL. */
		if (tid==k_tot-1)
		{
			endLoopIndex = loop_over_y_cells - 1;
			JL_d[N-1] = P*val_d[N-1]+Q*val_d[N-2]+R*(val_d[1]-2.0*val_d[N-1]+val_d[N-2]); /* For periodic integral */
		}
		else
			endLoopIndex = loop_over_y_cells;
		
		
		
		/* Perform the local weight step across the subdomain. 
		Every point in every subdomain (except for the domain boundary points) should be set here. */
        for(j=startLoopIndex;j<endLoopIndex; j++)

        {

            cell_index=j+tid*M;        /* Compute cell offset for cell j of */

                                       /* of sub domain domain tid */



                                       /* Compute integral */
			
			JL_d[cell_index]=P*val_d[cell_index]+Q*val_d[cell_index-1]+R*(val_d[cell_index+1]-2.0*val_d[cell_index]+val_d[cell_index-1]);
			//debugArray_d[cell_index] += 1;
        }
		
		
		/* Perform the local recursive sweep across the subdomain. */
        for(j=1; j<loop_over_y_cells; j++)

        {

            cell_index=j+tid*M;        /* Compute cell offset for cell j of */

                                       /* of sub domain domain tid */
			
			JL_d[cell_index]+=ex*JL_d[cell_index-1]; /* Perform recursive push */
			//debugArray_d[cell_index] += 1;

        }

        tid+=gridDim.x*blockDim.x;     /* jump to next sub domain this kernel */

                                       /* which is a whole cuda GRID away */

                                       /* grid dimension - gridDim.x*blockDim.x */

    };

}

/*
Function name: coarseSweep_L

Function type: CUDA kernel, but a serial process; should be called with one block/one thread

Purpose: performs the (serial) coarse grid sweep of the left integral part

Inputs: float * IL_d - pointer to array on device string of total integral value of the left part
		float * JL_d - pointer to array on device storing the local integral values of the left part
		const int N - number of grid points in the array
		const int M - number of grid points in one subdomain
		float nu - numerical parameter equal to dx*alpha
		int * debugArray_d - pointer to array of single-digit integers on device; used for diagnostic purposes only

Outputs: IL_d is written to on the coarse grid points, which for the left integral part are the last points of each subdomain, 
		with index of the form ((tid+1)*M-1). 
		At these points, these are the final integral values and need not be updated again.
*/

__global__ void coarseSweep_L(float * IL_d, float * JL_d, const int N, const int M, float nu, int *debugArray_d)
{

    int k = N/M;           /* number of whole sub domains */

    int k_end = N%M;         /* number of points in last domain */

    int k_tot;             /* total number of sub domains */


    int cell_index;     /* cells_index is the flattend index of the cell */

                           /* j is merely a counter */

    //bool test;             /* flag to indicate if we have a single nonuniform

                               //sub domain */
		


	float ex_subdom = exp(-nu*M);
	float ex_end = exp(-nu*k_end);


	float recursion_coeff;
	int subdom_offset;

    /* set up logic for when number of subdomains

     does not evenly divide total number of cells in a line. */

    if(k_end>0)

    {

        k_tot = k+1;       /* number of sub domains */

        //test = true;       /* flag for indicating a single special domain */

    }

    else

    {

        k_tot = k;         /* number of sub domains */

        //test = false;

    }



    /* use unique tid to identify which sub domain */

    int tid = threadIdx.x+blockIdx.x*blockDim.x;  //index of sub domain



    /* let the kernel address more than a single sub domain */

    while(tid<(k_tot-2)){

		if(tid==0){
			IL_d[M-1]=JL_d[M-1];
			//debugArray_d[M-1] += 1;
		}
		
		recursion_coeff = ex_subdom;
		subdom_offset = M;

        cell_index=M*(tid+1)-1;        /* Compute cell offset for cell j of */

                                       /* of sub domain domain tid */


			
		IL_d[cell_index+subdom_offset]=JL_d[cell_index+subdom_offset]+ IL_d[cell_index]*recursion_coeff;
		//debugArray_d[cell_index+subdom_offset] += 1;

        
		
        

        tid+=gridDim.x*blockDim.x;     /* jump to next sub domain this kernel */

                                       /* which is a whole cuda GRID away */

                                       /* grid dimension - gridDim.x*blockDim.x */

    };

}

/*
Function name: coarseToFineSweep_L

Function type: CUDA kernel

Purpose: performs the final coarse grid to fine grid sweep of the left integral part

Inputs: float * IL_d - pointer to array on device string of total integral value of the left part
		float * JL_d - pointer to array on device storing the local integral values of the left part
		const int N - number of grid points in the array
		const int M - number of grid points in one subdomain
		float nu - numerical parameter equal to dx*alpha
		int * debugArray_d - pointer to array of single-digit integers on device; used for diagnostic purposes only

Outputs: IL_d is written to everywhere in the domain, except for the endpoints which were already written to in coarseSweep_L
*/

__global__ void coarseToFineSweep_L(float * IL_d, float * JL_d, const int N, const int M, float nu, int * debugArray_d)
{

    int k = N/M;           /* number of whole sub domains */

    int k_end = N%M;         /* number of points in last domain */

    int k_tot;             /* total number of sub domains */

    int loop_over_y_cells; /* number of mesh point in domain */

    int cell_index,j;     /* cells_index is the flattend index of the cell */

                           /* j is merely a counter */

    bool test;             /* flag to indicate if we have a single nonuniform

                               sub domain */
		
	int startLoopIndex; /* Loop start index in subdomain. */
	int endLoopIndex; /* Loop end index. */
	int source_index;
	
	float ex = exp(-nu);



	float push_tracker;

    /* set up logic for when number of subdomains

     does not evenly divide total number of cells in a line. */

    if(k_end>0)

    {

        k_tot = k+1;       /* number of sub domains */

        test = true;       /* flag for indicating a single special domain */

    }

    else

    {

        k_tot = k;         /* number of sub domains */

        test = false;

    }



    /* use unique tid to identify which sub domain */

    int tid = threadIdx.x+blockIdx.x*blockDim.x;  //index of sub domain



    /* let the kernel address more than a single sub domain */

    while(tid<k_tot){
        if((tid==k_tot-1)&&(test))
		{
            loop_over_y_cells= k_end;

		}

        else
		{	
            loop_over_y_cells= M;
		
		}

		
        if(tid==k_tot-1)
		{
			source_index = M*(tid)-1;
			push_tracker = IL_d[source_index];
			startLoopIndex = 1;
			endLoopIndex = loop_over_y_cells+1;

		}
		else if(tid==0)
		{
			source_index = 0;
			push_tracker = (float)0;
			startLoopIndex = 1;
			endLoopIndex = loop_over_y_cells-1;
		}

        else
		{	
			source_index = M*(tid)-1;
			push_tracker = IL_d[source_index];
			startLoopIndex = 1;
			endLoopIndex = loop_over_y_cells;
		
		}



		
		/* Index of IL to be used as source for sweep - last grid point in subdom tid */
		//source_index = M*(tid+1)-1; 
		
		//push_tracker = IL_d[source_index];
		
        for(j=startLoopIndex;j<endLoopIndex; j++)

        {

            cell_index=source_index+j;        /* Compute cell offset for cell j of */

                                       /* of sub domain domain tid */

			
			IL_d[cell_index]= JL_d[cell_index]+ex*push_tracker;
			push_tracker = push_tracker*ex;
			
			//debugArray_d[cell_index] += 1;
        }

        tid+=gridDim.x*blockDim.x;     /* jump to next sub domain this kernel */

                                       /* which is a whole cuda GRID away */

                                       /* grid dimension - gridDim.x*blockDim.x */

    };

}


/*
Function name: localWeightAndSweep_R

Function type: CUDA kernel

Purpose: Performs the local deposit step then the local sweep step over each subdomain for the right integral part.

Inputs: float * JR_d - pointer to array on device storing the local integral values of the right part
		float * val_d - pointer to array on device storing the integrand values
		const int N - number of grid points in the array
		const int M - number of grid points in one subdomain
		float nu - numerical parameter equal to dx*alpha
		int * debugArray_d - pointer to array of single-digit integers on device; used for diagnostic purposes only

Outputs: JR_d is written to at all grid points
*/
__global__ void localWeightAndSweep_R(float * JR_d, float * val_d, const int N, const int M, float nu, int * debugArray_d)

{

    int k = N/M;           /* number of whole sub domains */

    int k_end = N%M;         /* number of points in last domain */

    int k_tot;             /* total number of sub domains */

    int loop_over_y_cells; /* number of mesh point in domain */

    int cell_index,j;     /* cells_index is the flattend index of the cell */

                           /* j is merely a counter */

    bool test;             /* flag to indicate if we have a single nonuniform

                               sub domain */
	
	int startLoopIndex; /* Loop start index in subdomain - should be 1 in first subdomain, 0 in others. */
	int endLoopIndex; /* Loop end index - should be k_end-1 in the last subdomain, M in others. */ 
		

	float ex = exp(-nu);

		/* Quadrature weights */
	float P = 1.0 - (1.0-ex)/nu;
	float Q = -ex+(1.0-ex)/nu;
	float R = (1.0-ex)/(nu*nu)-(1.0+ex)/(2.0*nu);



    /* set up logic for when number of subdomains

     does not evenly divide total number of cells in a line. */

    if(k_end>0)

    {

        k_tot = k+1;       /* number of sub domains */

        test = true;       /* flag for indicating a single special domain */

    }

    else

    {

        k_tot = k;         /* number of sub domains */

        test = false;

    }



    /* use unique tid to identify which sub domain */

    int tid = threadIdx.x+blockIdx.x*blockDim.x;  //index of sub domain



    /* let the krenal address more than a single sub domain */

    while(tid<k_tot){
		
        if((tid==k_tot-1)&&(test))

            loop_over_y_cells= k_end;

        else

            loop_over_y_cells= M;
		
		/* If we are in the first subdomain, the first point needs special treatment, as the usual quadrature stencil will extend outside of the domain. */
		/* We have that JR[0] is NOT zero, so we need special treatment of the left endpoint for JR. */
		if(tid==0)
		{
			startLoopIndex = 1;
			JR_d[0] = P*val_d[0]+Q*val_d[1]+R*(val_d[1]-2.0*val_d[0]+val_d[N-2]); /* For periodic integral */
		}
		else
		
			startLoopIndex = 0;

		
        

		
		/* If we are in the last subdomain, the last point needs special treatment, as the usual quadrature stencil will extend outside of the domain. */
		/* We have that JR[N-1]=0. */
		if (tid==k_tot-1)
		{
			endLoopIndex = loop_over_y_cells-1;
			JR_d[N-1]=0.0;
		}	
		else
			
			endLoopIndex = loop_over_y_cells;
		
		
		
		/* Perform the local weight step across the subdomain. */
        for(j=startLoopIndex;j<endLoopIndex; j++)

        {

            cell_index=tid*M+j;        /* Compute cell offset for cell j of */

                                       /* of sub domain domain tid */



                                       /* Compute integral */
			
			JR_d[cell_index]=P*val_d[cell_index]+Q*val_d[cell_index+1]+R*(val_d[cell_index+1]-2.0*val_d[cell_index]+val_d[cell_index-1]);
			//debugArray_d[cell_index] += 1;
        }
		
		
		/* Perform the local recursive sweep across the subdomain. */
        for(j=2;j<loop_over_y_cells+1; j++)

        {

            cell_index=tid*M+loop_over_y_cells-j;        /* Compute cell offset for cell j of */

                                       /* of sub domain domain tid */
			
			JR_d[cell_index]+=ex*JR_d[cell_index+1]; /* Perform recursive push */
			//debugArray_d[cell_index] += 1;

        }

        tid+=gridDim.x*blockDim.x;     /* jump to next sub domain this kernel */

                                       /* which is a whole cuda GRID away */

                                       /* grid dimension - gridDim.x*blockDim.x */

    };

}


/*
Function name: coarseSweep_R

Function type: CUDA kernel, but a serial process; should be called with one block/one thread

Purpose: performs the (serial) coarse grid sweep of the right integral part

Inputs: float * IR_d - pointer to array on device string of total integral value of the right part
		float * JR_d - pointer to array on device storing the local integral values of the right part
		const int N - number of grid points in the array
		const int M - number of grid points in one subdomain
		float nu - numerical parameter equal to dx*alpha
		int * debugArray_d - pointer to array of single-digit integers on device; used for diagnostic purposes only

Outputs: IR_d is written to on the coarse grid points, which for the right integral part are the first points of each subdomain, 
		with index of the form tid*M. 
		At these points, these are the final integral values and need not be updated again.
*/
__global__ void coarseSweep_R(float * IR_d, float * JR_d, const int N, const int M, float nu, int * debugArray_d)

{

    int k = N/M;           /* number of whole sub domains */

    int k_end = N%M;         /* number of points in last domain */

    int k_tot;             /* total number of sub domains */


    int cell_index;     /* cells_index is the flattend index of the cell */

                           /* j is merely a counter */

    bool test;             /* flag to indicate if we have a single nonuniform

                               sub domain */
	
	
	float ex_subdom = exp(-nu*M);
	float ex_end = exp(-nu*k_end);


	float recursion_coeff;
	int subdom_offset;

    /* set up logic for when number of subdomains

     does not evenly divide total number of cells in a line. */

    if(k_end>0)

    {

        k_tot = k+1;       /* number of sub domains */

        test = true;       /* flag for indicating a single special domain */

    }

    else

    {

        k_tot = k;         /* number of sub domains */

        test = false;

    }



    /* use unique tid to identify which sub domain */

    int tid = threadIdx.x+blockIdx.x*blockDim.x;  //index of sub domain



    /* let the kernel address more than a single sub domain */

    while(tid<k_tot-1){
		
		if(tid==0){
			if(test){
				recursion_coeff = ex_subdom;
				subdom_offset = M;
				cell_index = N-k_end;
			}
			else{
				recursion_coeff = ex_subdom;
				subdom_offset = M;
				cell_index = N-M;
			}
			IR_d[cell_index]=JR_d[cell_index];
			
			debugArray_d[cell_index] += 1;
		}
		else{
			if(test){
            	recursion_coeff = ex_subdom;
				subdom_offset = M;
				cell_index = N-k_end-tid*M;
			}	
			else{
				recursion_coeff = ex_subdom;
				subdom_offset = M;
				cell_index = N-(tid+1)*M;
			}
			//debugArray_d[cell_index] += 1;
		}

		

			
		IR_d[cell_index-subdom_offset]=JR_d[cell_index-subdom_offset] + IR_d[cell_index]*recursion_coeff;
		debugArray_d[cell_index-subdom_offset] += 1;

        
		
        

        tid+=gridDim.x*blockDim.x;     /* jump to next sub domain this kernel */

                                       /* which is a whole cuda GRID away */

                                       /* grid dimension - gridDim.x*blockDim.x */

    };


}


/*
Function name: coarseToFineSweep_R

Function type: CUDA kernel

Purpose: performs the final coarse grid to fine grid sweep of the right integral part

Inputs: float * IR_d - pointer to array on device string of total integral value of the right part
		float * JR_d - pointer to array on device storing the local integral values of the right part
		const int N - number of grid points in the array
		const int M - number of grid points in one subdomain
		float nu - numerical parameter equal to dx*alpha
		int * debugArray_d - pointer to array of single-digit integers on device; used for diagnostic purposes only

Outputs: IR_d is written to everywhere in the domain, except for the endpoints which were already written to in coarseSweep_R
*/
__global__ void coarseToFineSweep_R(float * IR_d, float * JR_d, const int N, const int M, float nu, int * debugArray_d)

{

    int k = N/M;           /* number of whole sub domains */

    int k_end = N%M;         /* number of points in last domain */

    int k_tot;             /* total number of sub domains */

    int loop_over_y_cells; /* number of mesh point in domain */

    int cell_index,j;     /* cells_index is the flattend index of the cell */

                           /* j is merely a counter */

    bool test;             /* flag to indicate if we have a single nonuniform

                               sub domain */
	
	int startLoopIndex; /* Loop start index in subdomain - should be 1 in first subdomain, 0 in others. */
	int endLoopIndex; /* Loop end index - should be k_end-1 in the last subdomain, M in others. */ 
	int source_index;


	float ex = exp(-nu);



	float push_tracker;

    /* set up logic for when number of subdomains

     does not evenly divide total number of cells in a line. */

    if(k_end>0)

    {

        k_tot = k+1;       /* number of sub domains */

        test = true;       /* flag for indicating a single special domain */

    }

    else

    {

        k_tot = k;         /* number of sub domains */

        test = false;

    }



    /* use unique tid to identify which sub domain */

    int tid = threadIdx.x+blockIdx.x*blockDim.x;  //index of sub domain



    /* let the kernel address more than a single sub domain */

    while(tid<k_tot){
        if((tid==k_tot-1)&&(test))
		{
            loop_over_y_cells= k_end;

		}

        else
		{	
            loop_over_y_cells= M;
		
		}

		
        if(tid==k_tot-1)
		{
			source_index = N-1;
			push_tracker = (float)0;
			startLoopIndex = 1;
			endLoopIndex = loop_over_y_cells-1;

		}

        else
		{	
			source_index = M*(tid+1);
			push_tracker = IR_d[source_index];
			startLoopIndex = 1;
			endLoopIndex = loop_over_y_cells;
		
		}
		
		
		
		
		
        for(j=startLoopIndex;j<endLoopIndex; j++)

        {

            cell_index=source_index-j;        /* Compute cell offset for cell j of */

                                       /* of sub domain domain tid */

			
			IR_d[cell_index]= JR_d[cell_index]+ex*push_tracker;
			push_tracker = push_tracker*ex;
			
			debugArray_d[cell_index] += 1;
        }

        tid+=gridDim.x*blockDim.x;     /* jump to next sub domain this kernel */

                                       /* which is a whole cuda GRID away */

                                       /* grid dimension - gridDim.x*blockDim.x */

    };

}

/*
Function name: vectorAdd

Function type: CUDA kernel

Purpose: adds two arrays on the device - I_d = IL_d + IR_d

Inputs: float * I_d - pointer to array on device containing the summed values
		float * IL_d - pointer to array on device
		float * IR_d - pointer to array on device
		const int N - number of grid points in the array
		const int M - number of grid points in one subdomain
		int * debugArray_d - pointer to array of single-digit integers on device; used for diagnostic purposes only
		
Outputs: I_d is written to at every grid point

*/

__global__ void vectorAdd(float * I_d, float * IL_d, float * IR_d, const int N, const int M, int * debugArray_d){

    int k = N/M;           /* number of whole sub domains */

    int k_end = N%M;         /* number of points in last domain */

    int k_tot;             /* total number of sub domains */

    int loop_over_y_cells; /* number of mesh point in domain */

    int cell_index,j;     /* cells_index is the flattend index of the cell */

                           /* j is merely a counter */

    bool test;             /* flag to indicate if we have a single nonuniform

                               sub domain */
	
	int startLoopIndex; /* Loop start index in subdomain - should be 1 in first subdomain, 0 in others. */
	int endLoopIndex; /* Loop end index - should be k_end-1 in the last subdomain, M in others. */ 



    /* set up logic for when number of subdomains

     does not evenly divide total number of cells in a line. */

    if(k_end>0)

    {

        k_tot = k+1;       /* number of sub domains */

        test = true;       /* flag for indicating a single special domain */

    }

    else

    {

        k_tot = k;         /* number of sub domians */

        test = false;

    }



    /* use uniuck tid to identify which sub domain */

    int tid = threadIdx.x+blockIdx.x*blockDim.x;  //index of sub domain



    /* let the krenal address more than a single sub domain */

    while(tid<k_tot){
		
		/* If we are in the first subdomain, the first point needs special treatment, as the usual quadrature stencil will extend outside of the domain. */
		/* We have that JL[0]=0. */

		
        if((tid==k_tot-1)&&(test))

            loop_over_y_cells= k_end;

        else

            loop_over_y_cells= M;


		startLoopIndex = 0;
		endLoopIndex = loop_over_y_cells;
		
		/* Perform the local weight step across the subdomain. */
        for(j=startLoopIndex;j<endLoopIndex; j++)

        {

            cell_index=j+tid*M;        /* Compute cell offset for cell j of */

                                       /* of sub domain domain tid */



                                       /* Compute integral */
			
			I_d[cell_index]=IL_d[cell_index]+IR_d[cell_index];
			
			
			//debugArray_d[cell_index] += 1;

        }
		
		
        tid+=gridDim.x*blockDim.x;     /* jump to next sub domain this kernel */

                                       /* which is a whole cuda GRID away */

                                       /* grid dimension - gridDim.x*blockDim.x */
    };

}



int main(void){

	clock_t start = clock(), setupTime, kernelTime, cleanupTime, testTime, diff;
	
	float L = 1.0;
	float dx = L/((float)(N-1));
	float nu = 1.0;
	float alpha = nu/dx;
	float x;
	float err = 0.0;
	float err_temp;
	
	int Nt = 1;
	
    float val[N];
	float I[N];    //host memory

	int debugArray[N];
	//int * debugArray_ptr = &debugArray;
	
	int * debugArray_d;

    float * val_d; // Integrand

	float * JL_d;
	float * JR_d;
	float * IL_d;
	float * IR_d;
	float * I_d;

    int num_B=1024,num_T=32;
	
	printf("Number of grid points: %i\n",N);
	
	printf("Number of time steps: %i\n",Nt);

    //allocate device memory

    hipMalloc((void **) &val_d,sizeof(float)*N);

	hipMalloc((void **) &JL_d,sizeof(float)*N);
	hipMalloc((void **) &JR_d,sizeof(float)*N);
	hipMalloc((void **) &IL_d,sizeof(float)*N);
	hipMalloc((void **) &IR_d,sizeof(float)*N);
	hipMalloc((void **) &I_d,sizeof(float)*N);

	hipMalloc((void **) &debugArray_d,sizeof(int)*N);
    //Set Inital Condtion...

    for(int i=0;i<N;i++){

        val[i]=(float) 1;

		I[i]=(float) 0;
		
		debugArray[i] = 0;
    };



    //mem copy


    hipMemcpy(val_d,val,sizeof(float)*N,hipMemcpyHostToDevice);
	hipMemcpy(JL_d,I,sizeof(float)*N,hipMemcpyHostToDevice);
	hipMemcpy(JR_d,I,sizeof(float)*N,hipMemcpyHostToDevice);
	hipMemcpy(IL_d,I,sizeof(float)*N,hipMemcpyHostToDevice);
	hipMemcpy(IR_d,I,sizeof(float)*N,hipMemcpyHostToDevice);
	hipMemcpy(I_d,I,sizeof(float)*N,hipMemcpyHostToDevice);
	
	hipMemcpy(debugArray_d,debugArray,sizeof(int)*N,hipMemcpyHostToDevice);

	setupTime = clock();
	diff = setupTime-start;
	int sec = diff/ CLOCKS_PER_SEC;
	printf("Setup time: %d seconds\n", sec);
	
    //Call kernel
	for(int n = 0; n<Nt; n++){
    localWeightAndSweep_L<<<num_B,num_T>>>(JL_d,val_d,N,M,nu,debugArray_d);
	coarseSweep_L<<<1,1>>>(IL_d,JL_d,N,M,nu,debugArray_d);
	coarseToFineSweep_L<<<num_B,num_T>>>(IL_d,JL_d,N,M,nu,debugArray_d);
    localWeightAndSweep_R<<<num_B,num_T>>>(JR_d,val_d,N,M,nu,debugArray_d);
	coarseSweep_R<<<1,1>>>(IR_d,JR_d,N,M,nu,debugArray_d);
	coarseToFineSweep_R<<<num_B,num_T>>>(IR_d,JR_d,N,M,nu,debugArray_d);
	vectorAdd<<<num_B,num_T>>>(I_d,IL_d,IR_d,N,M,debugArray_d);
	}
	
	kernelTime = clock();
	diff = kernelTime-setupTime;
	sec = diff/ CLOCKS_PER_SEC;
	printf("Kernel time: %d seconds\n", sec);

    //mem copy

    hipMemcpy(I,I_d,sizeof(float)*N,hipMemcpyDeviceToHost);
	hipMemcpy(debugArray,debugArray_d,sizeof(float)*N,hipMemcpyDeviceToHost);

    hipFree(val_d);
	hipFree(JL_d);
	hipFree(JR_d);
	hipFree(IL_d);
	hipFree(IR_d);
	hipFree(I_d);
	hipFree(debugArray_d);
	
	cleanupTime = clock();
	diff = cleanupTime-kernelTime;
	sec = diff/ CLOCKS_PER_SEC;
	printf("Clean up time: %d seconds\n",sec);
	
	/* Compute test integral value. */
	
	for(int j=0; j<N; j++){
		x = (float)j*dx;
		err_temp = abs(I[j]-(2.0-exp(-alpha*x)-exp(-alpha*(L-x))));
		
		if (err_temp>1.0e-6)
			printf("Error of %f at grid point j = %i \n", err_temp,j);
		
		if (err_temp>err){
			err = err_temp;
		};
		
	}
	
	printf("Maximum error: %f \n", err);
	
	testTime = clock();
	diff = testTime-cleanupTime;
	sec = diff/ CLOCKS_PER_SEC;
	printf("Test time: %d seconds\n",sec);
	

	//debug_tool_output_to_file( debugArray, N, M);
	
	return 0;


};